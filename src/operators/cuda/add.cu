
#include <hip/hip_runtime.h>
__global__
void add_kernel(int n, const float *x, const float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) z[i] = x[i] + y[i];
}

void add(int n, const float *x, const float *y, float *z) {
  add_kernel<<<(n+255)/256, 256>>>(n, x, y, z);
}
