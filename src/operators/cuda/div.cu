
#include <hip/hip_runtime.h>
__global__
void div_kernel(int n, const float *x, const float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) z[i] = x[i] / y[i];
}

void div(int n, const float *x, const float *y, float *z) {
  div_kernel<<<(n+255)/256, 256>>>(n, x, y, z);
}

