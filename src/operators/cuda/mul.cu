
#include <hip/hip_runtime.h>
__global__
void mul_kernel(int n, const float *x, const float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) z[i] = x[i] * y[i];
}

void mul(int n, const float *x, const float *y, float *z) {
  mul_kernel<<<(n+255)/256, 256>>>(n, x, y, z);
}
